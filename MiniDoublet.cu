#include "hip/hip_runtime.h"
# include "MiniDoublet.cuh"

void createMDsInUnifiedMemory(struct miniDoublets& mdsInGPU, unsigned int maxMDs, unsigned int nModules)
{
    hipMallocManaged(&mdsInGPU.hitIndices, maxMDs * 2 * sizeof(unsigned int));
    hipMallocManaged(&mdsInGPU.moduleIndices, maxMDs * sizeof(unsigned int));
    hipMallocManaged(&mdsInGPU.pixelModuleFlag, maxMDs * sizeof(short));
    hipMallocManaged(&mdsInGPU.dphichanges, maxMDs * sizeof(float));

    hipMallocManaged(&mdsInGPU.nMDs, nModules * sizeof(int));

    hipMallocManaged(&mdsInGPU.dzs, maxMDs * sizeof(float));
    hipMallocManaged(&mdsInGPU.dphis, maxMDs * sizeof(float));
    hipMallocManaged(&mdsInGPU.shiftedXs, maxMDs * sizeof(float));
    hipMallocManaged(&mdsInGPU.shiftedYs, maxMDs * sizeof(float));
    hipMallocManaged(&mdsInGPU.shiftedZs, maxMDs * sizeof(float));
    hipMallocManaged(&mdsInGPU.noShiftedDz, maxMDs * sizeof(float));
    hipMallocManaged(&mdsInGPU.noShiftedDphis, maxMDs * sizeof(float));
    hipMallocManaged(&mdsInGPU.noShiftedDphiChanges, maxMDs * sizeof(float));
}

void addMDToMemory(struct miniDoublets& mdsInGPU, struct hits& hitsInGPU, struct modules& modulesInGPU, unsigned int lowerHitIdx, unsigned int upperHitIdx, unsigned int lowerModuleIdx, float dz, float dphi, float dphichange, float shfitedX, float shiftedY, float shiftedZ, float noShiftedDz, float noShiftedDphi, float noShiftedDPhiChange, unsigned int idx)
{
    //the index into which this MD needs to be written will be computed in the kernel
    //nMDs variable will be incremented in the kernel, no need to worry about that here
    
    mdsInGPU.hitIndices[idx * 2] = lowerHitIdx;
    mdsInGPU.hitIndices[idx * 2 + 1] = upperHitIdx;
    mdsInGPU.moduleIndices[idx] = lowerModuleIdx;
    if(modulesInGPU.moduleType(lowerModuleIdx) == PS
    {
        if(modulesInGPU.moduleLayerType(lowerModuleIdx) == Pixel)
        {
            pixelModuleFlag = 0;
        }
        else
        {
            pixelModuleFlag = 1;
        }
    }
    else
    {
        pixelModuleFlag = -1;
    }

    mdsInGPU.dphichanges[idx] = dphichange;

    mdsInGPU.dphis[idx] = dphi;
    mdsInGPU.dzs[idx] = dz;
    mdsInGPU.shiftedXs[idx] = shiftedX;
    mdsInGPU.shiftedYs[idx] = shiftedY;
    mdsInGPU.shiftedZs[idx] = shiftedZ;

    mdsInGPU.noShiftedDzs[idx] = noshiftedDz;
    mdsInGPU.noShiftedDphis[idx] = noShiftedDphi;
    mdsInGPU.noShfitedDphiChanges[idx] = noShiftedDphiChange;
}

bool runMiniDoubletDefaultAlgoBarrel(struct modules& modulesInGPU, struct hits& hitsInGPU, float& dz, float& dPhi, float& dPhiChange, float& shiftedX, float& shiftedY, float& shiftedZ, float& noshiftedDz, float& noShiftedDphi, float& noShiftedDphichange)
{
}

bool runMiniDoubletDefaultAlgoEndcap(struct modules& modulesInGPU, struct hits& hitsInGPU, float& drt, float& dphi, float& dphichange, float& shiftedX, float& shiftedY, float& shiftedZ, float& noShiftedDz, float& noShiftedDphi, float& noShiftedDphiChange)
{

}

bool runMiniDoubletDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, float& dz, float& dphi, float& dphichange, float& shiftedX, float& shiftedY, float& shiftedZ, float& noShiftedDz, float& noShiftedDphi, float& noShiftedDphiChange)
{

}

float dPhiThreshold(struct hits& hitsInGPU, struct modules& modulesInGPU, unsigned int hitIndex, unsigned int moduleIndex)
{

}

inline float isTighterTiltedModules(struct modules& modulesInGPU, unsigned int moduleIndex)
{
    // The "tighter" tilted modules are the subset of tilted modules that have smaller spacing
    // This is the same as what was previously considered as"isNormalTiltedModules"
    // See Figure 9.1 of https://cds.cern.ch/record/2272264/files/CMS-TDR-014.pdf
    short subdet = modulesInGPU.subdets[moduleIndex];
    short layer = modulesInGPU.layers[moduleIndex];
    short side = modulesInGPU.sides[moduleIndex];
    short rod = modulesInGPU.rods[moduleIndex];

    if (
           (subdet == Barrel and side != Center and layer== 3)
           or (subdet == Barrel and side == NegZ and layer == 2 and rod > 5)
           or (subdet == Barrel and side == PosZ and layer == 2 and rod < 8)
           or (subdet == Barrel and side == NegZ and layer == 1 and rod > 9)
           or (subdet == Barrel and side == PosZ and layer == 1 and rod < 4)
       )
        return true;
    else
        return false;

}

inline float moduleGapSize(struct modules& modulesInGPU, unsigned int moduleIndex)
{
    float miniDeltaTilted[] = {0.26, 0.26, 0.26};
    float miniDeltaLooseTilted[] =  {0.4,0.4,0.4};
    float miniDeltaFlat[] =  {0.26, 0.16, 0.16, 0.18, 0.18, 0.18};
    float miniDeltaEndcap[5][15];

    for (size_t i = 0; i < 5; i++)
    {
        for (size_t j = 0; j < 15; j++)
        {
            if (i == 0 || i == 1)
            {
                if (j < 10)
                {
                    miniDeltaEndcap[i][j] = 0.4;
                }
                else
                {
                    miniDeltaEndcap[i][j] = 0.18;
                }
            }
            else if (i == 2 || i == 3)
            {
                if (j < 8)
                {
                    miniDeltaEndcap[i][j] = 0.4;
                }
                else
                {
                    miniDeltaEndcap[i][j]  = 0.18;
                }
            }
            else
            {
                if (j < 9)
                {
                    miniDeltaEndcap[i][j] = 0.4;
                }
                else
                {
                    miniDeltaEndcap[i][j] = 0.18;
                }
            }
        }
    }

    unsigned int iL = modulesInGPU.layers[moduleIndex]-1;
    unsigned int iR = modulesInGPU.rings[moduleIndex] - 1;
    short subdet = modulesInGPU.subdets[moduleIndex];
    short side = modulesInGPU.sides[moduleIndex];

    float moduleSeparation = 0;

    if (subdet == Barrel and side == Center)
    {
        moduleSeparation = miniDeltaFlat[iL];
    }
    else if (isTighterTiltedModules(modulesInGPU, moduleIndex))
    {
        moduleSeparation = miniDeltaTilted[iL];
    }
    else if (subdet == Endcap)
    {
        moduleSeparation = miniDeltaEndcap[iL][iR];
    }
    else //Loose tilted modules
    {
        moduleSeparation = miniDeltaLooseTilted[iL];
    }
}

void shiftStripHits(struct modules& modulesInGPU, struct hits& hitsInGPU, unsigned int lowerModuleIndex, unsigned int lowerHitIndex, unsigned int upperHitIndex, float* shiftedCoords)
{

    // This is the strip shift scheme that is explained in http://uaf-10.t2.ucsd.edu/~phchang/talks/PhilipChang20190607_SDL_Update.pdf (see backup slides)
    // The main feature of this shifting is that the strip hits are shifted to be "aligned" in the line of sight from interaction point to the the pixel hit.
    // (since pixel hit is well defined in 3-d)
    // The strip hit is shifted along the strip detector to be placed in a guessed position where we think they would have actually crossed
    // The size of the radial direction shift due to module separation gap is computed in "radial" size, while the shift is done along the actual strip orientation
    // This means that there may be very very subtle edge effects coming from whether the strip hit is center of the module or the at the edge of the module
    // But this should be relatively minor effect

    // dependent variables for this if statement
    // lowerModule
    // lowerHit
    // upperHit
    // SDL::endcapGeometry
    // SDL::tiltedGeometry

    // Some variables relevant to the function
    float xp; // pixel x (pixel hit x)
    float yp; // pixel y (pixel hit y)
    float xa; // "anchor" x (the anchor position on the strip module plane from pixel hit)
    float ya; // "anchor" y (the anchor position on the strip module plane from pixel hit)
    float xo; // old x (before the strip hit is moved up or down)
    float yo; // old y (before the strip hit is moved up or down)
    float xn; // new x (after the strip hit is moved up or down)
    float yn; // new y (after the strip hit is moved up or down)
    float abszn; // new z in absolute value
    float zn; // new z with the sign (+/-) accounted
    float angleA; // in r-z plane the theta of the pixel hit in polar coordinate is the angleA
    float angleB; // this is the angle of tilted module in r-z plane ("drdz"), for endcap this is 90 degrees
    bool isEndcap; // If endcap, drdz = infinity
    unsigned int pixelHitIndex; // Pointer to the pixel hit
    unsigned int stripHitIndex; // Pointer to the strip hit
    float moduleSeparation;
    float drprime; // The radial shift size in x-y plane projection
    float drprime_x; // x-component of drprime
    float drprime_y; // y-component of drprime
    float slope; // The slope of the possible strip hits for a given module in x-y plane
    float absArctanSlope;
    float angleM; // the angle M is the angle of rotation of the module in x-y plane if the possible strip hits are along the x-axis, then angleM = 0, and if the possible strip hits are along y-axis angleM = 90 degrees
    float absdzprime; // The distance between the two points after shifting
    float drdz_;

    // Assign hit pointers based on their hit type
    if (modulesInGPU.moduleType(lowerModuleIndex) == PS)
    {
        if (modulesInGPU.moduleLayerType(lowerModuleIndex)== Pixel)
        {
            pixelHitIndex = lowerHitIndex;
            stripHitIndex = upperHitIndex;
        }
        else
        {
            pixelHitIndex = upperHitIndex;
            stripHitIndex = lowerHitIndex;
        }
    }
    else // if (lowerModule.moduleType() == SDL::Module::TwoS) // If it is a TwoS module (if this is called likely an endcap module) then anchor the inner hit and shift the outer hit
    {
        pixelHitIndex = lowerHitIndex; // Even though in this case the "pixelHitPtr" is really just a strip hit, we pretend it is the anchoring pixel hit
        stripHitIndex = upperHitIndex;
    }

    // If it is endcap some of the math gets simplified (and also computers don't like infinities)
    isEndcap = modulesInGPU.subdets[lowerModuleIndex]== SDL::Module::Endcap;

    // NOTE: TODO: Keep in mind that the sin(atan) function can be simplifed to something like x / sqrt(1 + x^2) and similar for cos
    // I am not sure how slow sin, atan, cos, functions are in c++. If x / sqrt(1 + x^2) are faster change this later to reduce arithmetic computation time

    // The pixel hit is used to compute the angleA which is the theta in polar coordinate
    // angleA = std::atan(pixelHitPtr->rt() / pixelHitPtr->z() + (pixelHitPtr->z() < 0 ? M_PI : 0)); // Shift by pi if the z is negative so that the value of the angleA stays between 0 to pi and not -pi/2 to pi/2

    angleA = fabs(std::atan(hitsInGPU.rts[pixelHitIndex] / hitsInGPU.zs[pixelHitIndex]));
    // angleB = isEndcap ? M_PI / 2. : -std::atan(tiltedGeometry.getDrDz(detid) * (lowerModule.side() == SDL::Module::PosZ ? -1 : 1)); // The tilt module on the postive z-axis has negative drdz slope in r-z plane and vice versa
    drdz_ = modulesInGPU.drdzs[lowerModuleIndex];
    angleB = ((isEndcap) ? M_PI / 2. : atan(drdz_)); // The tilt module on the postive z-axis has negative drdz slope in r-z plane and vice versa


    moduleSeparation = moduleGapSize(modulesInGPU, moduleIndex);

    // Sign flips if the pixel is later layer
    if (modulesInGPU.moduleType(lowerModuleIndex) == PS and modulesInGPU.moduleLayerType(lowerModuleIndex) != Pixel)
    {
        moduleSeparation *= -1;
    }

    drprime = (moduleSeparation / std::sin(angleA + angleB)) * std::sin(angleA);
    slope = modulesInGPU.slopes[moduleIndex];

    // Compute arctan of the slope and take care of the slope = infinity case
    absArctanSlope = ((slope != SDL_INF) ? fabs(std::atan(slope)) : M_PI / 2); // Since C++ can't represent infinity, SDL_INF = 123456789 was used to represent infinity in the data table

    // The pixel hit position
    xp = hitsInGPU.xs[pixelHitIndex];
    yp = hitsInGPU.ys[pixelHitIndex];

    // Depending on which quadrant the pixel hit lies, we define the angleM by shifting them slightly differently
    if (xp > 0 and yp > 0)
    {
        angleM = absArctanSlope;
    }
    else if (xp > 0 and yp < 0)
    {
        angleM = M_PI - absArctanSlope;
    }
    else if (xp < 0 and yp < 0)
    {
        angleM = M_PI + absArctanSlope;
    }
    else // if (xp < 0 and yp > 0)
    {
        angleM = 2 * M_PI - absArctanSlope;
    }

    // Then since the angleM sign is taken care of properly
    drprime_x = drprime * std::sin(angleM);
    drprime_y = drprime * std::cos(angleM);

    // The new anchor position is
    xa = xp + drprime_x;
    ya = yp + drprime_y;

    // The original strip hit position
    xo = hitsInGPU.xs[stripHitIndex];
    yo = hitsInGPU.ys[stripHitIndex];

    // Compute the new strip hit position (if the slope vaule is in special condition take care of the exceptions)
    if (slope == SDL_INF) // Special value designated for tilted module when the slope is exactly infinity (module lying along y-axis)
    {
        xn = xa; // New x point is simply where the anchor is
        yn = yo; // No shift in y
    }
    else if (slope == 0)
    {
        xn = xo; // New x point is simply where the anchor is
        yn = ya; // No shift in y
    }
    else
    {
        xn = (slope * xa + (1.f / slope) * xo - ya + yo) / (slope + (1.f / slope)); // new xn
        yn = (xn - xa) * slope + ya; // new yn
    }

    // Computing new Z position
    absdzprime = fabs(moduleSeparation / std::sin(angleA + angleB) * std::cos(angleA)); // module separation sign is for shifting in radial direction for z-axis direction take care of the sign later

    // Depending on which one as closer to the interactin point compute the new z wrt to the pixel properly
    if (lowerModule.moduleLayerType() == SDL::Module::Pixel)
    {
        abszn = std::abs(hitsInGPU.zs[pixelHitIndex]) + absdzprime;
    }
    else
    {
        abszn = std::abs(hitsInGPU.zs[pixelHitIndex]) - absdzprime;
    }

    zn = abszn * ((hitsInGPU.zs[pixelHitIndex] > 0) ? 1 : -1); // Apply the sign of the zn

/*    if (logLevel == SDL::Log_Debug3)
    {
        SDL::cout << upperHit << std::endl;
        SDL::cout << lowerHit << std::endl;
        SDL::cout <<  " lowerModule.moduleType()==SDL::Module::PS: " << (lowerModule.moduleType()==SDL::Module::PS) <<  std::endl;
        SDL::cout <<  " lowerModule.moduleLayerType()==SDL::Module::Pixel: " << (lowerModule.moduleLayerType()==SDL::Module::Pixel) <<  std::endl;
        SDL::cout <<  " pixelHitPtr: " << pixelHitPtr <<  std::endl;
        SDL::cout <<  " stripHitPtr: " << stripHitPtr <<  std::endl;
        SDL::cout <<  " detid: " << detid <<  std::endl;
        SDL::cout <<  " isEndcap: " << isEndcap <<  std::endl;
        SDL::cout <<  " pixelHitPtr->rt(): " << pixelHitPtr->rt() <<  std::endl;
        SDL::cout <<  " pixelHitPtr->z(): " << pixelHitPtr->z() <<  std::endl;
        SDL::cout <<  " angleA: " << angleA <<  std::endl;
        SDL::cout <<  " angleB: " << angleB <<  std::endl;
        SDL::cout <<  " moduleSeparation: " << moduleSeparation <<  std::endl;
        SDL::cout <<  " drprime: " << drprime <<  std::endl;
        SDL::cout <<  " slope: " << slope <<  std::endl;
        SDL::cout <<  " absArctanSlope: " << absArctanSlope <<  std::endl;
        SDL::cout <<  " angleM: " << angleM <<  std::endl;
        SDL::cout <<  " drprime_x: " << drprime_x <<  std::endl;
        SDL::cout <<  " drprime_y: " << drprime_y <<  std::endl;
        SDL::cout <<  " xa: " << xa <<  std::endl;
        SDL::cout <<  " ya: " << ya <<  std::endl;
        SDL::cout <<  " xo: " << xo <<  std::endl;
        SDL::cout <<  " yo: " << yo <<  std::endl;
        SDL::cout <<  " xn: " << xn <<  std::endl;
        SDL::cout <<  " yn: " << yn <<  std::endl;
        SDL::cout <<  " absdzprime: " << absdzprime <<  std::endl;
        SDL::cout <<  " zn: " << zn <<  std::endl;
    }*/

    shiftedCoords[0] = xn;
    shiftedCoords[1] = yn;
    shiftedCoords[2] = zn;

}


